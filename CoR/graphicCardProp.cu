/**
*  Copyright � [2011], Empa, Juergen Hofmann
*/

#include "typesFDK.h"
#include "recon.h"
#include <vector>
#include <iostream>

using namespace std;

extern "C"
void GetGraphicCardProp(vector<GraphicCardProp> &gpuPropV)
{
	// get graphic card property
	////////////////////////////
	// get number of available graphic devices
	const unsigned MB = 1024*1024;
	int cnt;
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceCount(&cnt));
	for(int i = 0; i < cnt; i++)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&prop,i));
		GraphicCardProp gpuProp;
  		gpuProp.globalMem		= prop.totalGlobalMem/MB;
		gpuProp.maxThrPerBlk	= prop.maxThreadsPerBlock;
		gpuProp.majorRev		= prop.major;
		gpuProp.minorRev		= prop.minor;
		gpuProp.multProcCnt		= prop.multiProcessorCount ;
		gpuPropV.push_back(gpuProp);
	}
}