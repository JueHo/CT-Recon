#include "hip/hip_runtime.h"
/**
*  Copyright © [2011], Empa, Juergen Hofmann
*/

#include "reconFDK_kernelCoR.cuh"
#include "recon.h"
#include "typesFDK.h"
#include <cfloat>
#include <climits>
#include <algorithm>
#include <cstdlib>
#include "hip/hip_runtime_api.h"
#include "reader.h"
#include <cstdio>

extern char *parafilename;

extern int gMultiRowFlag;
extern int gCurrentSlice;


extern QVector<double> g_X;
extern QVector<double> g_Y;

//-ju- vector for linear square fit
extern std::vector<float> h_val;
extern std::vector<float> h_slice;



#include "rot_ax_det.h"
using namespace cv;


//-ju-19-Jan-2011 set PRINT_INFO to 1 to print out add. information
///////////////////////////////////////////////////////////////////
#define PRINT_INFO 0

//const size_t FILE_NUMBER_LEN=5;

inline float ROUND(float x)
{
    return x > 0.0f ? x + 0.5f : x - 0.5f;
}

extern "C"
    int runFDK(BackProjParam param, int devNo, float *minGray, float *maxGray, FILE *fpVol, PlotData *plData)
{
    int numberDevice, device;
    hipGetDeviceCount(&numberDevice);
    if (numberDevice > 0) 
    {
        int maxMultiprocessors = 0, maxDevice = 0, maxMajor = 0;
        string deviceName = "none";
        for (device = 0; device < numberDevice; device++) 
        {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, device);
            if (maxMajor <= prop.major && maxMultiprocessors <= prop.multiProcessorCount) 
            {
                maxMultiprocessors = prop.multiProcessorCount;
                maxMajor = prop.major;
                maxDevice = device;
                deviceName = prop.name;
            }
        }
        if(gMultiRowFlag == 0)
        {
            printf("\nSelect GPU device number: %d\nDevice name: %s\n\n",maxDevice,deviceName.c_str());
        }
        hipSetDevice(maxDevice);
    }
    hipSetDevice(0);

    clock_t start, stop, io_start, io_stop;
    float io_time = 0;
    start = clock();


    // estimate number of chunks of z-slices
    // depending on available memory
    int number_y_chunks = static_cast<int>(param.sliceChunkSize);
    // stacksize of z-slices process as a block at once
    int backproj_chunks	= param.volY/number_y_chunks;
    const int backproj_chunks_remainder = param.volY%number_y_chunks;
    bool hasYChunkRemainder = false;
    if(backproj_chunks_remainder != 0)
    {
        number_y_chunks += 1;
        hasYChunkRemainder = true;
        if(gMultiRowFlag == 0)
        {
            cout << "Y Chunk Remainder is true\n\n";
        }
    }

    // number of projections processed at once (depends on gpu memory)
    // number of projection blocks depends on avail. graphic card memory
    int proj_blcks = static_cast<int>(param.projectionChunkSize); 
    int nProjUse = param.numProj/proj_blcks;    // number of projections to use
    int remainProj = param.numProj%proj_blcks;
    bool hasRemainder = false;
    // if has remainder
    if(remainProj != 0)
    {
        proj_blcks += 1;
        hasRemainder = true;
        if(gMultiRowFlag == 0)
        {
            cout << "Projection Chunk Remainder is true\n\n";
        }
    }


    FDK_Constants *tmp_c= (FDK_Constants*)malloc(sizeof(FDK_Constants));
    tmp_c->imgWidth		= param.imgWidth;
    tmp_c->imgHeight	= param.imgHeight;
    tmp_c->numProj		= param.numProj;
    tmp_c->projProcSize = nProjUse;
    tmp_c->angleIncr	= 2.0f*PI/param.numProj;
    tmp_c->voxSize		= param.binFac*param.pixSize/param.magnification; // pixel-size / magnification
    tmp_c->distSrcObj	= param.dso;
    tmp_c->offsetX = (float)(param.imgWidth)/2.0f - 0.5f;
    tmp_c->offsetY = (float)(param.imgHeight)/2.0f - 0.5f;

#if PRINT_INFO == 1
    cout << "Offset x: " << tmp_c->offsetX << endl;
    cout << "Offset y: " << tmp_c->offsetY << endl;
#endif

    if(gMultiRowFlag == 0)
    {
        cout << "Volume dimensions: [" << param.volX << "," << param.volY << "," << param.volZ << "]\n";
        cout.flush();
    }

    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(fdkConst), tmp_c, sizeof(FDK_Constants)));
    free(tmp_c);

    unsigned int dimThreadsX = param.threadX;
    unsigned int dimThreadsY = param.threadY;
    dim3 threads(dimThreadsX,dimThreadsY);

    // blocks in grid
     int dimBlocksX, dimBlocksZ;
    dimBlocksX = (param.volX  + threads.x - 1)/threads.x;
    dimBlocksZ = (param.volZ + threads.y - 1)/threads.y;
    dim3 blocks(dimBlocksX,dimBlocksZ);

    // min and max values in tomograms
    // used for scaling from float to integer
    float tomoMax = FLT_MIN;
    float tomoMin = FLT_MAX;

    //-ju-25-May-2015 out put to file calculating rotation axis
    ////////////////////////////////////////////////////////////
    FILE *varMax;
    char varMaxFile[256];
    sprintf(varMaxFile,"%s.rotaxis",parafilename);
    varMax = fopen(varMaxFile,"w");
    if(varMax == 0)
    {
        std::cerr << "Can not open calc. rotation axis out put file:" << varMaxFile << endl;
        exit(-346);
    }

    FILE *excelIn;
    char excelInFile[256];
    sprintf(excelInFile,"%s.rot_axis.ExcelIn",parafilename);
    excelIn = fopen(excelInFile,"w");
    if(varMax == 0)
    {
        std::cerr << "Can not open calc. rotation axis excel input file:" << varMaxFile << endl;
        exit(-347);
    }

    //-ju-25-May-2015 out put to file calculating rotation axis
    ////////////////////////////////////////////////////////////
    int indexSh         = 0;
    int indexShSD       = 0;
	float maxVariance	= 0.0f;
	float minVariance	= FLT_MAX;
    float maxStddev     = 0.0f;
    float maxShift      = 0.0f;
    float maxShiftSD    = 0.0f;

    for(int nck = 0; nck < number_y_chunks; nck++)
    {
        // dynamic z chunk size
        // if last chunk and has remainder
        int yChunkEnd;
        int yChunkStart;
        if(nck == number_y_chunks-1 && hasYChunkRemainder)
        {
            // save start condition before change backproj_chunks
            yChunkStart = nck*backproj_chunks;
            backproj_chunks = backproj_chunks_remainder;
            yChunkEnd = param.volY;
        }
        else
        {
            yChunkStart = nck*backproj_chunks;
            yChunkEnd = (nck+1)*backproj_chunks;
        }

        if(gMultiRowFlag == 0)
        {
            printf("Reconstruction of slices: %4d - %4d  of total %d  slices\n", 
                yChunkStart+1, yChunkEnd, param.volY);
            fflush(stdout);
        }
        //-ju- allocate memory for a chunk of sinograms
        unsigned long long sinChunkSize = (size_t)param.imgWidth*(size_t)param.imgHeight*(size_t)nProjUse;
        float *h_sinoData = new float[sinChunkSize]();

        /*-ju-27-Oct-2011 new */
        unsigned long long chunkSize = (unsigned long long)param.volX*(unsigned long long)param.volZ*(unsigned long long)backproj_chunks;
        float *h_backproj_chunk = new float[chunkSize];


        for(int np=0; np<proj_blcks; np++) // projection blocks
        {
            //-ju-13-Dec-2010 add projection remainder handling
            int start_value;
            int end_value;
            if(hasRemainder && np == proj_blcks-1)
            {
                start_value = np*nProjUse;
                end_value = np*nProjUse + remainProj;
            }
            else
            {
                start_value = np*nProjUse;
                end_value = (np+1)*nProjUse;
            }

            if(gMultiRowFlag == 0)
            {
                printf(" processing: projection block %2d of %2d projection blocks\n", np+1, proj_blcks);
                fflush(stdout);
            }
            int   cuProjBlockIdx	= np; // current projection block index

            // read chunk of sinograms
            //////////////////////////
            io_start = clock();
            int rval;
            if(param.useFileList == 1)
            {
                if((rval = ReadChunkOfSinogramsFromFileList(start_value,
                    end_value,
                    param,
                    &h_sinoData[0]))!=0)
                {
                    fprintf(stderr,"\n\t==> Error reading chunk of sinograms!\n\n");
                    return rval;
                }
            }
            else
            {
                if(param.sino32BlockUse)
                {
                    if((rval = ReadBlockChunkOfSinograms(start_value,
                        end_value,
                        param,
                        &h_sinoData[0]))!=0)
                    {
                        fprintf(stderr,"\n\t==> Error reading block chunk of sinograms!\n\n");
                        return rval;
                    }
                }
                else
                {
                    if((rval = ReadChunkOfSinograms(start_value,
                        end_value,param,
                        &h_sinoData[0]))!=0)
                    {
                        fprintf(stderr,"\n\t==> Error reading chunk of sinograms!\n\n");
                        return rval;
                    }
                }
            }
            io_stop = clock();
            io_time += (float)(io_stop - io_start) / CLOCKS_PER_SEC;
			
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
            //-ju- device memory of sinogram data
            hipArray *d_sino;
            //-ju- allocate memory holding texture device memory
	    	const hipExtent volumeSize = make_hipExtent((size_t)param.imgWidth, (size_t)param.imgHeight, (size_t)nProjUse);
            checkCudaErrors(hipMalloc3DArray(&d_sino, &channelDesc, volumeSize, hipArrayLayered));
            //-ju- copy memory host to device
            hipMemcpy3DParms copyParams = {0};
			copyParams.srcPos = make_hipPos((size_t)0, (size_t)0, (size_t)0);
			copyParams.dstPos = make_hipPos((size_t)0, (size_t)0, (size_t)0);
			copyParams.extent = make_hipExtent((size_t)param.imgWidth, (size_t)param.imgHeight, (size_t)nProjUse);
            copyParams.kind     = hipMemcpyHostToDevice;
            copyParams.dstArray = d_sino;
            copyParams.srcPtr   = make_hipPitchedPtr(h_sinoData,
                volumeSize.width*sizeof(float),
                volumeSize.width,
                volumeSize.height);
            checkCudaErrors(hipMemcpy3D(&copyParams));
			
	    	hipTextureObject_t tex3DLayObj = 0;
			hipResourceDesc texRes;
			memset(&texRes, 0, sizeof(hipResourceDesc));

			texRes.resType = hipResourceTypeArray;
			texRes.res.array.array = d_sino;

			hipTextureDesc texDescr;
			memset(&texDescr, 0, sizeof(hipTextureDesc));

			texDescr.normalizedCoords = false;
			texDescr.filterMode = hipFilterModeLinear;
			texDescr.addressMode[0] = hipAddressModeBorder;
			texDescr.addressMode[1] = hipAddressModeBorder;
			texDescr.readMode = hipReadModeElementType;

			checkCudaErrors(hipCreateTextureObject(&tex3DLayObj, &texRes, &texDescr, NULL));

            int index = 0;
            for(int iy=yChunkStart; iy<yChunkEnd; iy++)
            {
                //-ju- allocate memory for one backprojection slice on device
                float *d_backProj;
                checkCudaErrors(hipMalloc( (void**)&d_backProj, param.volX*param.volZ*sizeof(float)));

                unsigned long long offset = (unsigned long long)param.volX*(unsigned long long)param.volZ*(unsigned long long)index;
                checkCudaErrors(hipMemcpy(d_backProj,
                    (char*)&h_backproj_chunk[offset], 
                    param.volX*param.volZ*sizeof(float),
                    hipMemcpyHostToDevice));

                //////////////////////////////////
				//-ju- Run Backprojection Kernel
                ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            
				// using a "one slice size" block ==> index of it is 0
				int   y_c = 0; 


                float horizShift = (float(iy)+(float)param.pixShiftMin)*param.shiftStep;

                //-ju- Run Backprojection Kernel
                /////////////////////////////////
                 if(hasRemainder && np == proj_blcks-1) // process remainder
                {
                    fdk_kernel_3DW_R_HA<<<blocks,threads>>>(d_backProj, y_c, 
                        cuProjBlockIdx, remainProj, 
                        param.volX, param.volZ, 
                        horizShift, param.shiftY, 
                        param.wVolOrigX, param.wVolOrigY, param.wVolOrigZ,
                        tex3DLayObj);
                }
                else
                {
                    //-ju-05-03-2014 standard kernel
                    ////////////////////////////////
                    fdk_kernel_3DW_HA<<<blocks,threads>>>(d_backProj, y_c, 
                        cuProjBlockIdx, 
                        param.volX, param.volZ, 
                        horizShift, param.shiftY, 
                        param.wVolOrigX, param.wVolOrigY, param.wVolOrigZ,
                        tex3DLayObj);
                }

                getLastCudaError("Kernel execution failed");

                checkCudaErrors(hipMemcpy((char*)&h_backproj_chunk[offset], d_backProj, (size_t)param.volX*param.volZ*sizeof(float), hipMemcpyDeviceToHost));
                //-ju- free memory for one z-slice
                checkCudaErrors(hipFree(d_backProj));
                index++;
            }

            //-ju- free device memory used for sinograms (chunks wise)
			checkCudaErrors(hipDestroyTextureObject(tex3DLayObj));
            checkCudaErrors(hipFreeArray(d_sino));
        }


        // Write tomgrams
        //////////////////
        io_start = clock();
        ofstream s_backProjFile;
        int index = 0;
		if (!CreateTomoDir(param.tomoDir.c_str()))
		{
            cout << "Can not create >> " << param.tomoDir << endl;
            cout.flush();
            exit(-3);
        };
        int startIdx = param.startIndex;
        char tomgramFile[500];


		double bilat_time_start, bilat_time_stop, bilat_sum = 0.0;
        for(int i=yChunkStart; i<yChunkEnd; i++)
        {
            // offset in memory block
            unsigned long long offset = (unsigned long long)param.volX*(unsigned long long)param.volZ*(unsigned long long)index;
            //-ju-19-Oct-2011 calc. min/max
            unsigned long long endSeq = offset + (unsigned long long)param.volX*(unsigned long long)param.volZ;
            for(unsigned long long  k = offset; k < endSeq; k++)
            {
                if(h_backproj_chunk[k] < tomoMin)
                    tomoMin = h_backproj_chunk[k];
                if(h_backproj_chunk[k] > tomoMax)
                    tomoMax = h_backproj_chunk[k];
            }

            long long ii = i;
            // file offset
            unsigned long long fileOffset = 
                ((unsigned long long)param.volY - (unsigned long long)(ii+startIdx))*(unsigned long long)param.volX*(unsigned long long)param.volZ;
            if(param.tomoBlockWrite)
            {
                _fseeki64(fpVol,fileOffset*sizeof(float),0);
                fwrite((char*)&h_backproj_chunk[offset], param.volX*param.volZ*sizeof(float), 1, fpVol);
            }
            else
            {
                //-ju-14-Apr-2015 --> add
                int id;
				//-ju-25-may-2015 for rotation axis determination  
				// reverse projection stack orientation is not implemented
				// (to be done ???)
                if(1 == 1/*param.viewDirection*/) 
                {
                    id = i+startIdx;
                }
                else
                {
                    id = param.volY - (i+startIdx) + 1;
                }
                //-ju-14-Apr-2015 <-- add
                 if(gMultiRowFlag == 0)
                {
                    sprintf(tomgramFile,"%s%s%04dx%04dx%04d_%04d.raw", 
                        param.tomoDir.c_str(),param.tomoName.c_str(),
                        param.volX,param.volY,param.volZ,
                        id);
                    s_backProjFile.open(tomgramFile, ios::binary);
                    if(!s_backProjFile.is_open())
                    {
                        cout << "Can not open " << tomgramFile << "  -  Exit program!\n";
                        cout.flush();
                        return -1;
                    }
                    if(!s_backProjFile.write((char*)&h_backproj_chunk[offset], param.volX*param.volZ*sizeof(float)))
                    {
                        cout << "\nError writing tomogram\n";
                        cout.flush();
                        return -2;
                    }
                s_backProjFile.close();
                }

                //-ju-25-May-2015 calculation for rotation axis determination
                /////////////////////////////////////////////////////////////
                float horizShift = (float(i)+(float)param.pixShiftMin)*param.shiftStep;
                //-ju-28-may-2015 in memory calculation Mat orig, var, stddev;
                Mat var, stddev;
                Scalar var_sum, stddev_sum;
                unsigned w = param.volX;
                unsigned h = param.volZ;
                //-ju-28-may-2015 in memory calculation
                ////////////////////////////////////////
                Mat orig = Mat::zeros(h, w, CV_32F);
                memcpy(orig.data, &h_backproj_chunk[offset], param.volX*param.volZ*sizeof(float));
				bilat_time_start = clock();
				
				//////////////////////////////////////
				//-ju-22-Jan-2017 actual used filter
				/////////////////////////////////////
				// Run Sharpness Filter
				//////////////////////////////////////////////////////////////////////////////////////
				GradientSquaredFromBiLat(tomgramFile, w, h, orig, var, var_sum, stddev, stddev_sum);//
				//////////////////////////////////////////////////////////////////////////////////////

				bilat_time_stop = clock();
				bilat_sum += (float)(bilat_time_stop - bilat_time_start) / CLOCKS_PER_SEC;
                if(gMultiRowFlag == 0)
                {
                    printf("Index=%+4d\tShift=%+7.4f\tSharpness=%+8.5f\t\n",i,horizShift,var_sum.val[0]);
                    fprintf(varMax,"%+10d %+10.3f %+10.6f\n",i,horizShift,var_sum.val[0]);
                    fprintf(excelIn,"%+10.3f %+10.6f\n",horizShift,var_sum.val[0]);
                    g_X.push_back(horizShift);
                    g_Y.push_back(var_sum.val[0]);
                }
				// ju-08-07-2015
				if (var_sum.val[0]>maxVariance)
                {
                    maxVariance = var_sum.val[0];
                    indexSh   = i;
                    maxShift = horizShift;
                }
				
                if(stddev_sum.val[0]>maxStddev)
                {
                    maxStddev = stddev_sum.val[0];
                    indexShSD   = i;
                    maxShiftSD  = horizShift;
                }
            }
            index++;
        }
#ifdef _PRINT_OUT
		printf("\n\n\t--->Filtering takes %f s time\n\n", bilat_sum);
#endif

        io_stop = clock();
        io_time += (float)(io_stop - io_start) / CLOCKS_PER_SEC;

        delete [] h_backproj_chunk;
        delete [] h_sinoData;
    }


    if(gMultiRowFlag == 0)
    {
		//-ju-08-07-2015
		printf("\n\n\nMaximum:\n\tSlice No=%d\th-shift=%f\tsharp coef=%f\n", indexSh + 1, maxShift, maxVariance);
		plData->maxYVal = maxVariance;
		plData->maxXVal = maxShift;
	}
    else
    {
		printf("     Maximum:\tat image slice no=%d\th-shift=%f\tsharp coef=%f\n\n", indexSh + 1, maxShift, maxVariance);
		//-ju-linear square fit
		h_val.push_back(maxShift);
		h_slice.push_back((float)(indexSh + 1)); //
	}
	fprintf(varMax, "\n\n\nSharpness (best value):\nslice no=%d\n\th-shift=%f\n\tvalue=%f\n", indexSh + 1, maxShift, minVariance);
    fclose(varMax);
    fclose(excelIn);


     //-ju-21-Oct-2014 hipDeviceReset(); --> replaced by hipDeviceReset()
    hipDeviceReset();
    stop = clock();
    float duration = (float)(stop - start) / CLOCKS_PER_SEC;
    *maxGray = tomoMax;
    *minGray = tomoMin;


    size_t pos = param.projectName.find(".");
    string scalename = param.projectName.substr(0,pos);
    scalename += ".log";

    if(gMultiRowFlag == 0)
    {

        printf("\n--------------Timming----------------------\n");
        printf("Total runtime:              %#4.1f seconds\n", duration );
        printf("IO time without conversion: %#4.1f seconds\n", io_time );
        printf("Runtime without io:         %#4.1f seconds\n", duration-io_time);
        printf("Minimal gray value:         %f\n", tomoMin);
        printf("Maximal gray value:         %f\n", tomoMax);
        printf("---------------------------------------------\n");
    }
    FILE *fp;
    fp = fopen(scalename.c_str(),"w");
    if(fp == 0)
    {
        printf("Can not open for writing: %s\n", scalename.c_str());
        return -1;
    }
    fprintf(fp,"\n---------------------------------------------\n");
    fprintf(fp,"Total runtime:              %#4.1f seconds\n", duration );
    fprintf(fp,"IO time without conversion: %#4.1f seconds\n", io_time );
    fprintf(fp,"Runtime without io:         %#4.1f seconds\n", duration-io_time);
    fprintf(fp,"Tomo datatype:				%s\n", param.outDataType.c_str());
    fprintf(fp,"Tomo width:					%4d\n", param.volX);
    fprintf(fp,"Tomo depth:					%4d\n", param.volZ);
    fprintf(fp,"Tomo height:				%4d\n", param.volY);
    fprintf(fp,"Minimal gray value:         %f\n", tomoMin);
    fprintf(fp,"Maximal gray value:         %f\n", tomoMax);
    fprintf(fp,"---------------------------------------------\n");

    fclose(fp);

    return 0;
}


