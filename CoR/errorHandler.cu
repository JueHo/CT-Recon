/**
*  Copyright � [2011], Empa, Juergen Hofmann
*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

extern "C"
void HandleError(hipError_t err,const char *file,int line) 
{
    if (err != hipSuccess) 
	{
        printf( "%s in %s at line %d\n",hipGetErrorString(err),file, line );
        exit(EXIT_FAILURE);
    }
}
