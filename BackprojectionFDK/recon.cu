#include "hip/hip_runtime.h"
/**
*  Copyright � [2011], Empa, Juergen Hofmann
*/

#include "reconFDK_kernel.cuh"
#include "recon.h"
#include "typesFDK.h"
#include <cfloat>
#include <climits>
#include <algorithm>
#include <cstdlib>
#include "hip/hip_runtime_api.h"
#include "reader.h"
#include <cstdio>

#include <omp.h>

//-ju-19-Jan-2011 set PRINT_INFO to 1 to print out add. information
///////////////////////////////////////////////////////////////////
#define PRINT_INFO 0

//const size_t FILE_NUMBER_LEN=5;

inline float ROUND(float x)
{
	return x > 0.0f ? x + 0.5f : x - 0.5f;
}

// algorithms used in transform
///////////////////////////////
class RangeMapI16
{
public:
	RangeMapI16(float mapFactor, float f32min)
		:m_mapFactor(mapFactor),m_f32min(f32min){}
	short operator() (float inVal)
	{
		return static_cast<short>(ROUND((inVal - m_f32min)*m_mapFactor + SHRT_MIN));
	}
private:
	float m_mapFactor;
	float m_f32min;
};

class RangeMapUI16
{
public:
	RangeMapUI16(float mapFactor, float f32min)
		:m_mapFactor(mapFactor),m_f32min(f32min){}
	unsigned short operator() (float inVal)
	{
		return static_cast<unsigned short>(ROUND((inVal - m_f32min)*m_mapFactor));
	}
private:
	float m_mapFactor;
	float m_f32min;
};

class RangeMapUC8
{
public:
	RangeMapUC8(float mapFactor, float f32min)
		:m_mapFactor(mapFactor),m_f32min(f32min){}
	unsigned char operator() (float inVal)
	{
		return static_cast<unsigned char>(ROUND((inVal - m_f32min)*m_mapFactor));
	}
private:
	float m_mapFactor;
	float m_f32min;
};


void InitAngleTable(const BackProjParam &param)
{
	if(param.angleList.size() > MAX_PROJ_NUM)
	{
		cout << "Number of angle table entries (" 
			 << param.angleList.size() 
			 << ") exceeds pre-defined max.size of " 
			 << MAX_PROJ_NUM << endl;
		exit(-2);
	}
	vector<float> rads_temp(param.angleList.size());
	float angle2rad = 2.0f*PI/360.0f;
	for(int i = 0; i < rads_temp.size(); i++)
	{
		rads_temp[i] = param.angleList[i]*angle2rad;
	}
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(radsConst), &rads_temp[0], sizeof(float)*rads_temp.size()));
}


extern "C"
int runFDK(BackProjParam param, int devNo, float *minGray, float *maxGray, FILE *fpVol)
{
    //hipSetDevice(0);
    //-ju-19-Nov-2013 automatically select device with highest performance 
    //                 ==> max. number of multiprocessors
    int numberDevice, device;
    hipGetDeviceCount(&numberDevice);
    if (numberDevice > 0) 
    {
        int maxMultiprocessors = 0, maxDevice = 0, maxMajor = 0;
        string deviceName = "none";
        for (device = 0; device < numberDevice; device++) 
        {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, device);
            if (maxMajor <= prop.major && maxMultiprocessors <= prop.multiProcessorCount) 
            {
                maxMultiprocessors = prop.multiProcessorCount;
                maxMajor = prop.major;
                maxDevice = device;
                deviceName = prop.name;
            }
        }
        printf("\nSelect GPU device number: %d\nDevice name: %s\n\n",maxDevice,deviceName.c_str());
		hipSetDevice(maxDevice);
    }
	//hipSetDevice(0);

    clock_t start, stop, io_start, io_stop;
	float io_time = 0;
	start = clock();

	if(param.useFileList == 1)
	{
		InitAngleTable(param);
	}

	// estimate number of chunks of z-slices
	// depending on available memory
	//int number_z_chunks = 17;  // hard coded
	int number_y_chunks = static_cast<int>(param.sliceChunkSize);
	// stacksize of z-slices process as a block at once
	int backproj_chunks	= param.volY/number_y_chunks;
	const int backproj_chunks_remainder = param.volY%number_y_chunks;
	bool hasYChunkRemainder = false;
	if(backproj_chunks_remainder != 0)
	{
		number_y_chunks += 1;
		hasYChunkRemainder = true;
		cout << "Y Chunk Remainder is true\n\n";
	}

	// number of projections processed at once (depends on gpu memory)
	// number of projection blocks depends on avail. graphic card memory
	int proj_blcks = static_cast<int>(param.projectionChunkSize);
	int nProjUse = param.numProj/proj_blcks;    // number of projections to use
	int remainProj = param.numProj%proj_blcks;
	bool hasRemainder = false;
	// if has remainder
	if(remainProj != 0)
	{
		proj_blcks += 1;
		hasRemainder = true;
		cout << "Projection Chunk Remainder is true\n\n";
	}


	FDK_Constants *tmp_c= (FDK_Constants*)malloc(sizeof(FDK_Constants));
	tmp_c->imgWidth		= param.imgWidth;
	tmp_c->imgHeight	= param.imgHeight;
	tmp_c->numProj		= param.numProj;
	tmp_c->projProcSize = nProjUse;
	tmp_c->angleIncr	= 2.0f*PI/param.numProj;
	tmp_c->voxSize		= param.binFac*param.pixSize/param.magnification; // pixel-size / magnification
	tmp_c->distSrcObj	= param.dso;
	tmp_c->offsetX = (float)(param.imgWidth)/2.0f - 0.5f;
	tmp_c->offsetY = (float)(param.imgHeight)/2.0f - 0.5f;

#if PRINT_INFO == 1
	cout << "Offset x: " << tmp_c->offsetX << endl;
	cout << "Offset y: " << tmp_c->offsetY << endl;
#endif

	cout << "Volume dimensions: [" << param.volX << "," << param.volY << "," << param.volZ << "]\n";
	cout.flush();

	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(fdkConst), tmp_c, sizeof(FDK_Constants)));
	free(tmp_c);

	unsigned int dimThreadsX = param.threadX;
	unsigned int dimThreadsY = param.threadY;
	dim3 threads(dimThreadsX,dimThreadsY);

	// blocks in grid
	//int dimBlocksX, dimBlocksY;
	int dimBlocksX, dimBlocksZ;
	dimBlocksX = (param.volX  + threads.x - 1)/threads.x;
	dimBlocksZ = (param.volZ + threads.y - 1)/threads.y;
	dim3 blocks(dimBlocksX,dimBlocksZ);

	// min and max values in tomograms
	// used for scaling from float to integer
	float tomoMax = FLT_MIN;
	float tomoMin = FLT_MAX;

	for(int nck = 0; nck < number_y_chunks; nck++)
	{
		// dynamic z chunk size
		// if last chunk and has remainder
		int yChunkEnd;
		int yChunkStart;
		if(nck == number_y_chunks-1 && hasYChunkRemainder)
		{
			// save start condtion before change backproj_chunks
			yChunkStart = nck*backproj_chunks;
			backproj_chunks = backproj_chunks_remainder;
			yChunkEnd = param.volY;
		}
		else
		{
			yChunkStart = nck*backproj_chunks;
			yChunkEnd = (nck+1)*backproj_chunks;
		}

		printf("Reconstruction of slices: %4d - %4d  of total %d  slices\n", 
			yChunkStart+1, yChunkEnd, param.volY);
		fflush(stdout);

		//-ju- allocate memory for a chunk of sinograms
		unsigned long long sinChunkSize = (size_t)param.imgWidth*(size_t)param.imgHeight*(size_t)nProjUse;
		float *h_sinoData = new float[sinChunkSize]();
		//-ju- allocate memory for backprojection for one slice on host 

		/*-ju-27-Oct-2011 new */
		unsigned long long chunkSize = (unsigned long long)param.volX*(unsigned long long)param.volZ*(unsigned long long)backproj_chunks;
#ifdef _ASYNC_IO
		float *h_backproj_chunk;
		chunkSize = chunkSize*(unsigned long long)sizeof(float);
		HANDLE_ERROR(hipHostMalloc((void**)&h_backproj_chunk, chunkSize));
#else
		float *h_backproj_chunk = new float[chunkSize];
#endif

		for(int np=0; np<proj_blcks; np++) // projection blocks
		{
			//-ju-13-Dec-2010 add projection remainder handling
			int start_value;
			int end_value;
			if(hasRemainder && np == proj_blcks-1)
			{
				start_value = np*nProjUse;
				end_value = np*nProjUse + remainProj;
				//nProjUse = remainProj;
			}
			else
			{
				start_value = np*nProjUse;
				end_value = (np+1)*nProjUse;
			}

			printf(" processing: projection block %2d of %2d projection blocks\n", np+1, proj_blcks);
			fflush(stdout);
			int   cuProjBlockIdx	= np; // current projection block index

			// read chunk of sinograms
			//////////////////////////
			io_start = clock();
			int rval;
			if(param.useFileList == 1)
			{
				if((rval = ReadChunkOfSinogramsFromFileList(start_value,
					                                        end_value,
															param,
															&h_sinoData[0]))!=0)
				{
					fprintf(stderr,"\n\t==> Error reading chunk of sinograms!\n\n");
					return rval;
				}
			}
			else
			{
				if(param.sino32BlockUse)
				{
					if((rval = ReadBlockChunkOfSinograms(start_value,
														 end_value,
														 param,
														 &h_sinoData[0]))!=0)
					{
						fprintf(stderr,"\n\t==> Error reading block chunk of sinograms!\n\n");
						return rval;
					}
				}
				else
				{
					if((rval = ReadChunkOfSinograms(start_value,
						                            end_value,param,
													&h_sinoData[0]))!=0)
					{
						fprintf(stderr,"\n\t==> Error reading chunk of sinograms!\n\n");
						return rval;
					}
#if 0
					char buf[64];
					sprintf(buf,"Block%04d_%04d.buf",np,proj_blcks);
					std::ofstream blkwrite(buf,std::ios::binary);
					if(blkwrite.is_open())
					{
						blkwrite.write((char*) &h_sinoData[0],(size_t)param.imgWidth*(size_t)param.imgHeight*(size_t)nProjUse*4); 
						blkwrite.close();
					}
#endif
				}
			}
			io_stop = clock();
			io_time += (float)(io_stop - io_start) / CLOCKS_PER_SEC;
			hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
			//-ju- device memory sinogram
			hipArray *d_sino;
			//-ju- allocate memory holding texture device memory
			const hipExtent volumeSize = make_hipExtent(param.imgWidth, param.imgHeight, nProjUse);
			//HANDLE_ERROR(hipMalloc3DArray(&d_sino, &channelDesc, volumeSize));
			HANDLE_ERROR(hipMalloc3DArray(&d_sino, &channelDesc, volumeSize, hipArrayLayered));
			//-ju- copy memory host to device
			hipMemcpy3DParms copyParams = {0};
			copyParams.srcPos   = make_hipPos(0,0,0);
			copyParams.dstPos   = make_hipPos(0,0,0);
			copyParams.extent   = make_hipExtent(param.imgWidth, param.imgHeight, nProjUse);
			copyParams.kind     = hipMemcpyHostToDevice;
			copyParams.dstArray = d_sino;
			copyParams.srcPtr   = make_hipPitchedPtr(h_sinoData,
													  volumeSize.width*sizeof(float),
													  volumeSize.width,
													  volumeSize.height);
			HANDLE_ERROR(hipMemcpy3D(&copyParams));

			//-ju- channel descriptor
			//hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
			//texSino.addressMode[0]     = hipAddressModeClamp;
			texSinoLay.addressMode[0]     = hipAddressModeBorder; //-ju- -> set out of border access to 0
			//texSino.addressMode[1]     = hipAddressModeClamp;
			texSinoLay.addressMode[1]     = hipAddressModeBorder;
			texSinoLay.filterMode         = hipFilterModeLinear;
            //texSino.filterMode         = hipFilterModePoint;
			texSinoLay.normalized         = false;

			//-ju- bind texture to array device memory
			HANDLE_ERROR(hipBindTextureToArray(texSinoLay, d_sino, channelDesc));

			//-ju-11-Dec-2015 add streams
			hipStream_t *stream = (hipStream_t *)malloc(STREAM_SIZE * sizeof(hipStream_t));
			for (int i = 0; i < STREAM_SIZE; i++)
			{
				hipStreamCreate(&(stream[i]));
			}


			int index = 0;
			////float *d_backProj[STREAM_SIZE];
			////for (int i = 0; i < STREAM_SIZE; i++)
			////{
			////	HANDLE_ERROR(hipMalloc((void**)&d_backProj[i], MAP*param.volX*param.volZ*sizeof(float)));
			////}
			//for(int iy=yChunkStart; iy<yChunkEnd; iy++)
			for (int iy = yChunkStart; iy<yChunkEnd; iy = iy + MAP*STREAM_SIZE)
			{
				// Allocate and initialize an array of stream handles
				//-ju- allocate memory for one backprojection slice on device
				float *d_backProj[STREAM_SIZE];
				for (int i = 0; i < STREAM_SIZE; i++)
				{
					HANDLE_ERROR(hipMalloc((void**)&d_backProj[i], MAP*param.volX*param.volZ*sizeof(float)));
				}

				//-ju-11-Dec-2015 loop over stream
				//////////////////////////////////
				unsigned long long offset[STREAM_SIZE];
				for (int istm = 0; istm < STREAM_SIZE; istm++)
				{
					//-ju- (re-)copy host memory for backprojection to device
					offset[istm] = (unsigned long long)param.volX*(unsigned long long)param.volZ*(unsigned long long)(index + istm*MAP);
					hipMemcpyAsync(d_backProj[istm], 
									(char*)&h_backproj_chunk[offset[istm]], 
									MAP*param.volX*param.volZ*sizeof(float), 
									hipMemcpyHostToDevice, 
									stream[istm]);

					//-ju- Run Backprojection Kernel
					/////////////////////////////////
					int   y_c = iy + istm*MAP;     // current z slice

					float offset_current = param.shiftX;

					if (param.applyTiltCorr == 1)
					{
						// Values from regression
						// y = mx + c
						float m = static_cast<float>(param.slopeTiltCorr);
						float c = static_cast<float>(param.constTiltCorr);
						offset_current = m * static_cast<float>(y_c) + c;
						if ((y_c % 100) == 0)
						{
							std::cout << "Using horizontal offset=" << offset_current << " at slice=" << y_c << std::endl;
						}
					}


					// 3D-Window volume selection reconstruction
					if (param.useVolWin)
					{
						if (hasRemainder && np == proj_blcks - 1) // process remainder
						{
							//-ju-03-Nov switch between fast and high acc. bi-linear interpolation
							///////////////////////////////////////////////////////////////////////
							if (param.highAcc == 0)
							{
								fdk_kernel_3DW_R << <blocks, threads >> >(d_backProj[istm], y_c,
									cuProjBlockIdx, remainProj,
									param.volX, param.volZ,
									offset_current, param.shiftY,
									param.wVolOrigX, param.wVolOrigY, param.wVolOrigZ);
							}
							else
							{
								fdk_kernel_3DW_R_HA << <blocks, threads >> >(d_backProj[istm], y_c,
									cuProjBlockIdx, remainProj,
									param.volX, param.volZ,
									offset_current, param.shiftY,
									param.wVolOrigX, param.wVolOrigY, param.wVolOrigZ);
							}
						}
						else
						{
							//-ju-03-Nov switch between fast and high acc. bi-linear interpolation
							///////////////////////////////////////////////////////////////////////
							if (param.highAcc == 0)
							{
								//-ju-05-03-2014 standard kernel
								fdk_kernel_3DW << <blocks, threads >> >(d_backProj[istm], y_c,
									cuProjBlockIdx,
									param.volX, param.volZ,
									offset_current, param.shiftY,
									param.wVolOrigX, param.wVolOrigY, param.wVolOrigZ);
							}
							else
							{
								fdk_kernel_3DW_HA << <blocks, threads >> >(d_backProj[istm], y_c,
									cuProjBlockIdx,
									param.volX, param.volZ,
									offset_current, param.shiftY,
									param.wVolOrigX, param.wVolOrigY, param.wVolOrigZ);
							}
						}
					}
					getLastCudaError("Kernel execution failed");

					hipMemcpyAsync((char*)&h_backproj_chunk[offset[istm]],
									d_backProj[istm],
						            MAP*param.volX*param.volZ*sizeof(float),
									hipMemcpyDeviceToHost,
						            stream[istm]);

				} //-ju-11-Dec-2015 streams end

				for (int i = 0; i < STREAM_SIZE; i++)
				{
					HANDLE_ERROR(hipFree(d_backProj[i]));
				}
				index = index + MAP*STREAM_SIZE;

			}
			////for (int i = 0; i < STREAM_SIZE; i++)
			////{
			////	HANDLE_ERROR(hipFree(d_backProj[i]));
			////}
			//-ju-11-Dec-2015 add streams
			// release all stream
			for (int i = 0; i < STREAM_SIZE; i++)
			{
				hipStreamDestroy(stream[i]);
			}

			//-ju- unbind texture
			hipUnbindTexture(texSinoLay);

			//-ju- free device memory used for sinograms (chunks wise)
			HANDLE_ERROR(hipFreeArray(d_sino));

		}

		// Write tomgrams
		//////////////////
		io_start = clock();
		ofstream s_backProjFile;
		int index = 0;
		if(!CreateTomoDir(param.tomoDir.c_str()))
		{
			cout << "Can not create >> " << param.tomoDir << endl;
			cout.flush();
			exit(-3);
		};
		int startIdx = param.startIndex;
		char tomgramFile[500];
        
		//-ju-18-Dec-2015 add openmp
//		omp_set_num_threads(2);
//#pragma omp parallel for shared(h_backproj_chunk)
		for(int i =yChunkStart; i<yChunkEnd; i++)
		{
			// offset in memory block
			unsigned long long offset = (unsigned long long)param.volX*(unsigned long long)param.volZ*(unsigned long long)index;
			//-ju-19-Oct-2011 calc. min/max
			unsigned long long endSeq = offset + (unsigned long long)param.volX*(unsigned long long)param.volZ;
			for(unsigned long long  k = offset; k < endSeq; k++)
			{
				if(h_backproj_chunk[k] < tomoMin)
					tomoMin = h_backproj_chunk[k];
				if(h_backproj_chunk[k] > tomoMax)
					tomoMax = h_backproj_chunk[k];
			}

			long long ii = i;
			// file offset
			unsigned long long fileOffset = 
				((unsigned long long)param.volY - (unsigned long long)(ii+startIdx))*(unsigned long long)param.volX*(unsigned long long)param.volZ;
			if(param.tomoBlockWrite)
			{
				_fseeki64(fpVol,fileOffset*sizeof(float),0);
				fwrite((char*)&h_backproj_chunk[offset], param.volX*param.volZ*sizeof(float), 1, fpVol);
				//index++;
			}
			else
			{
                //-ju-14-Apr-2015 --> add
                int id;
                if(1 == param.viewDirection)
                {
                    id = i+startIdx;
                    //printf("DPC Write id=%d\n", id);
                }
                else
                {
                    id = param.volY - (i+startIdx) + 1;
                    //printf("ABS Write id=%d\n", id);
                }
                //-ju-14-Apr-2015 <-- add
				//-ju-14-Apr-2015 int id = param.volY - (i+startIdx) + 1;
				sprintf(tomgramFile,"%s%s%04dx%04dx%04d_%04d.raw", 
					param.tomoDir.c_str(),param.tomoName.c_str(),
					param.volX,param.volZ,param.volY,
					id);
				s_backProjFile.open(tomgramFile, ios::binary);
				if(!s_backProjFile.is_open())
				{
					cout << "Can not open " << tomgramFile << "  -  Exit program!\n";
					cout.flush();
					return -1;
				}
				if(!s_backProjFile.write((char*)&h_backproj_chunk[offset], param.volX*param.volZ*sizeof(float)))
				{
					cout << "\nError writing tomogram\n";
					cout.flush();
					return -2;
				}
				s_backProjFile.close();
			}
			index++;
		}
        
		io_stop = clock();
		io_time += (float)(io_stop - io_start) / CLOCKS_PER_SEC;

#ifdef _ASYNC_IO
		hipHostFree(h_backproj_chunk);
#else
		delete[] h_backproj_chunk;
#endif
		delete[] h_sinoData;
		//-ju-27-Oct-2011delete [] h_backProj;
	}
	//-ju-15-Mar-2011 release resources lookup
	//ReleaseLookUpResources();

	//-ju-21-Oct-2014 hipDeviceReset(); --> replaced by hipDeviceReset()
    hipDeviceReset();
	stop = clock();
	float duration = (float)(stop - start) / CLOCKS_PER_SEC;
	*maxGray = tomoMax;
	*minGray = tomoMin;

	/*
	if(param.outDataType == "byte")
	{
		printf("\n\n--------------------------------------------\n");
		printf("  Converting Real 32-Bit to 8-Bit tomgrams\n");
		ConvertTomoDataUC8(param, tomoMax, tomoMin);
		printf("--------------------------------------------\n");
	}
	if(param.outDataType == "short16")
	{
		printf("\n\n----------------------------------------------------\n");
		printf("  Converting Real 32-Bit to Signed 16-Bit tomgrams\n");
		ConvertTomoDataI16(param, tomoMax, tomoMin);
		printf("----------------------------------------------------\n");
	}
	if(param.outDataType == "ushort16")
	{
		printf("\n\n----------------------------------------------------\n");
		printf("  Converting Real 32-Bit to unsigned 16-Bit tomgrams\n");
		ConvertTomoDataUI16(param, tomoMax, tomoMin);
		printf("----------------------------------------------------\n");
	}
	if(param.outDataType == "real32")
	{
		// do nothing
	}
    */

	size_t pos = param.projectName.find(".");
	string scalename = param.projectName.substr(0,pos);
	scalename += ".log";

	printf("\n--------------Timming----------------------\n");
	printf("Total runtime:              %#4.1f seconds\n", duration );
	printf("IO time without conversion: %#4.1f seconds\n", io_time );
	printf("Runtime without io:         %#4.1f seconds\n", duration-io_time);
	printf("Minimal gray value:         %f\n", tomoMin);
	printf("Maximal gray value:         %f\n", tomoMax);
	printf("---------------------------------------------\n");

	FILE *fp;
	fp = fopen(scalename.c_str(),"w");
	if(fp == 0)
	{
		printf("Can not open for writing: %s\n", scalename.c_str());
		return -1;
	}
	fprintf(fp,"\n---------------------------------------------\n");
	fprintf(fp,"Total runtime:              %#4.1f seconds\n", duration );
	fprintf(fp,"IO time without conversion: %#4.1f seconds\n", io_time );
	fprintf(fp,"Runtime without io:         %#4.1f seconds\n", duration-io_time);
	fprintf(fp,"Tomo datatype:				%s\n", param.outDataType.c_str());
	fprintf(fp,"Tomo width:					%4d\n", param.volX);
	fprintf(fp,"Tomo depth:					%4d\n", param.volZ);
	fprintf(fp,"Tomo height:				%4d\n", param.volY);
	fprintf(fp,"Minimal gray value:         %f\n", tomoMin);
	fprintf(fp,"Maximal gray value:         %f\n", tomoMax);
	fprintf(fp,"---------------------------------------------\n");

	fclose(fp);

	return 0;
}
