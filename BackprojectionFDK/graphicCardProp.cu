/**
*  Copyright � [2011], Empa, Juergen Hofmann
*/
/**********************************************************
*
* $Log: graphicCardProp.cu $
* Revision 1.2  2010/12/21 06:47:32  Hofmann
* remove not used keywords
*
* Revision 1.1  2010/12/15 09:52:49  Hofmann
* Initial revision
*
**********************************************************/
#include "typesFDK.h"
#include "recon.h"
#include <vector>
#include <iostream>

using namespace std;

extern "C"
void GetGraphicCardProp(vector<GraphicCardProp> &gpuPropV)
{
	// get graphic card property
	////////////////////////////
	// get number of available graphic devices
	const unsigned MB = 1024*1024;
	int cnt;
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceCount(&cnt));
	for(int i = 0; i < cnt; i++)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&prop,i));
		GraphicCardProp gpuProp;
        /*
        gpuProp.name = prop.name;

        //cout << "\n\n ---->>>> " << prop.name << "\n\n";

        basic_string <char>::size_type indexCh;
        const char *cstr = "Tesla";
        indexCh = gpuProp.name.find(cstr,0);
        if (indexCh != string::npos)

        if(prop.tccDriver) gpuProp.tccDriver = true;
        */
		gpuProp.globalMem		= prop.totalGlobalMem/MB;
		gpuProp.maxThrPerBlk	= prop.maxThreadsPerBlock;
		gpuProp.majorRev		= prop.major;
		gpuProp.minorRev		= prop.minor;
		gpuProp.multProcCnt		= prop.multiProcessorCount ;
		gpuPropV.push_back(gpuProp);
	}
}