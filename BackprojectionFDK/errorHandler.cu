/**
*  Copyright � [2011], Empa, Juergen Hofmann
*/
/**********************************************************

*
* $Log: errorHandler.cu $
* Revision 1.1  2010/12/15 09:52:49  Hofmann
* Initial revision
*
*
**********************************************************/

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

extern "C"
void HandleError(hipError_t err,const char *file,int line) 
{
    if (err != hipSuccess) 
	{
        printf( "%s in %s at line %d\n",hipGetErrorString(err),file, line );
        exit(EXIT_FAILURE);
    }
}
